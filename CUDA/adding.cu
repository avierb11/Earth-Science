
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void addKernel(float *a, float *b, int length)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;

  if (id < length) a[id] += b[id];
}

void printArray(float *arr, int length)
{
    for (int i = 0; i < length; i++)
    {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}

__global__ void testKernel(float *heads, float *queue, int length)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;

  if (id < length) queue[id] = .3f;
}

__global__ void queueKernel(float *heads, float *queue, int length)
{
  int id = blockDim.x*blockIdx.x + threadIdx.x;
  int final = length - 1;

  // Check to make sure it's not either end
  if ((id != 0) && (id != final))
  {
      queue[id] =  .1*(heads[id + 1] - heads[id]);
      queue[id] += .1*(heads[id - 1] - heads[id]);
  } else if (id == 0)
  {
    queue[0] = 1.0;
    queue[0] = .1*(heads[1] - heads[0]);
  } else if (id == final)
  {
    queue[final] = .1*(heads[final-1]-heads[final]);
  }
}

void getQueue(float *heads, float *queue, int length, int iters)
{
  int final = length - 1;
  bool debug = false;
  const int num = length*sizeof(float);
  // Create device variables
  float *h, *q;
  // Allocate memory
  hipMalloc(&h, num);
  hipMalloc(&q, num);
  // Copy to device
  hipMemcpy(h, heads, num, hipMemcpyHostToDevice);
  hipMemcpy(q, queue, num, hipMemcpyHostToDevice);
  // Run kernels
  for (int i = 0; i < iters; i++)
  {
    queueKernel<<<length,1>>> (h, q, length);
    addKernel<<<length,1>>> (h,q,length);
  }
  // Copy back
  hipMemcpy(heads, h, num, hipMemcpyDeviceToHost);
  hipMemcpy(queue, q, num, hipMemcpyDeviceToHost);
  hipFree(h);
  hipFree(q);

}

void add(float *a, float *b, int length)
{
  const int mem = length*sizeof(float);
  float *d_a, *d_b;

  // Allocate memory
  hipMalloc(&d_a,mem);
  hipMalloc(&d_b,mem);

  hipMemcpy(d_a,a,mem,hipMemcpyHostToDevice);
  hipMemcpy(d_b,b,mem,hipMemcpyHostToDevice);

  addKernel<<<1,length>>>(d_a,d_b,length);

  hipMemcpy(b,d_b,mem,hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);

}

int main(void)
{
  const int length = 5;
  const int mem = length*sizeof(mem);

  float *heads, *queue;

  heads = (float*)malloc(mem);
  queue = (float*)malloc(mem);

  for (int i = 0; i < length; i++)
  {
    heads[i] = 0.0f;
    queue[i] = 0.0f;
  }
  heads[0] = 1.0f;

  getQueue(heads,queue,length, 1);
  std::cout<<"heads: ";
  printArray(heads,length);


  free(heads);
  free(queue);

  return 0;
}
