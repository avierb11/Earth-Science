

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

using namespace std::chrono;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y, float *z)
{
int index = threadIdx.x;
int stride = blockDim.x;
for (int i = index; i < n; i += stride)
    z[i] = x[i]*y[i]*.001;
}


__global__
void add2D(int m, int n, float *x[4], float *y[4], float *z[4])
{
  int indexX = threadIdx.x;
  int strideX = blockDim.x;
  int indexY = threadIdx.y;
  int strideY = blockDim.y;

  for (int i = indexY; i < m; i += strideY)
  {
    for (int j = indexX; i < n; i += strideX)
    {
      z[i][j] = x[i][j] + y[i][j];
    }
  }
}

void printArray(float arr[4][4]);

void addNaive(int n, float *x, float *y, float *z)
{
  for (int i = 0; i < n; i++)
  {
    z[i] = x[i]*y[i]*.001;
  }
}

int main(void)
{
//int blocks = 1024*2;
//int threads = 1024;
int N = 1<<22;
float *x, *y, *z;

int smallRange = 4;
float (*smallX)[4];
float (*smallY)[4];
float (*smallZ)[4];

for (int i = 0; i < smallRange; i++)
{
  for (int j = 0; j < smallRange; j++)
  {
    smallX[i][j] = 1.0f;
    smallY[i][j] = 2.0f;
  }
}


// Allocate Unified Memory – accessible from CPU or GPU
hipMallocManaged(&x, N*sizeof(float));
hipMallocManaged(&y, N*sizeof(float));
hipMallocManaged(&z, N*sizeof(float));

// initialize x and y arrays on the host
for (int i = 0; i < N; i++) {
  x[i] = 1.0f;
  y[i] = 2.0f;
}

// run first function
auto start0 = high_resolution_clock::now();
for (int i = 0; i < 50; i++)
{
  addNaive(N, x, y, z);
}
auto stop0 = high_resolution_clock::now();



auto duration0 = duration_cast<microseconds>(stop0 - start0);
//std::cout << duration0.count() << " for CPU function" << std::endl;

// Reset values
for (int i = 0; i < N; i++) {
  x[i] = 1.0f;
  y[i] = 2.0f;
}
auto start = high_resolution_clock::now();
// Run kernel on 1M elements on the GPU
int count = 0;
for (int i = 0; i < 50; i++)
{
  add<<<4, 1024>>>(N, x, y, z);
  count++;
}
// Wait for GPU to finish before accessing on host
hipDeviceSynchronize();
auto stop = high_resolution_clock::now();

//std::cout << "count: " << count << std::endl;


// Check for errors (all values should be 3.0f)

float maxError = 0.0f;
for (int i = 0; i < N; i++)
  maxError = fmax(maxError, fabs(z[i]-3.0f));
//std::cout << "Max error: " << maxError << std::endl;

auto duration = duration_cast<microseconds>(stop - start);
float time = (float)duration.count()/1000000;
//std::cout << duration.count()<< " for GPU function" << std::endl;

//std::cout << time << " seconds" << std::endl;
// Free memory
hipFree(x);
hipFree(y);

float multiplier;
multiplier = (float)duration0.count()/(float)duration.count();

//std::cout << "GPU computing is " << multiplier << " times faster" << std::endl;
std::cout << "made it almost to the end" << std::endl;

printArray(smallX);

return 0;
}

void printArray(float arr[4][4])
{
  for (int i = 0; i < 4; i++)
  {
    for (int j = 0; j < 4; j++)
    {
      std::cout << arr[i][j] << " ";
    }
    std::cout << std::endl;
  }
}
